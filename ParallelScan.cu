#include <iostream>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>


typedef unsigned int uint;

#define MAX_THREADS_PER_BLOCK 1024
#define MAX_BLOCKS 65535

using namespace std;


#define CUDA_CHECK(val) checkCudaError((val), #val, __FILE__, __LINE__)

////////////////////////////////////////////////////////////////////////////////////////////
// Define utility function to check CUDA errors
void checkCudaError(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result != hipSuccess) {
            cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n" <<
            "CUDA error message = " << hipGetErrorString(result) << "\n";
        exit(1);
    }
}
////////////////////////////////////////////////////////////////////////////////////////////
hipError_t allocateDeviceMemory(uint** devicePtr, size_t size) {
    hipError_t err = hipMalloc((void**)devicePtr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return err;
}
////////////////////////////////////////////////////////////////////////////////////////////
hipError_t copyToDevice(uint* devicePtr, const uint* hostPtr, size_t size) {
    hipError_t err = hipMemcpy(devicePtr, hostPtr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy memory to the device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return err;
}
////////////////////////////////////////////////////////////////////////////////////////////
hipError_t copyFromDevice(uint* hostPtr, const uint* devicePtr, size_t size) {
    hipError_t err = hipMemcpy(hostPtr, devicePtr, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy memory from the device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return err;
}
////////////////////////////////////////////////////////////////////////////////////////////
hipError_t deallocateDeviceMemory(uint* devicePtr) {
    hipError_t err = hipFree(devicePtr);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device memory (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return err;
}
////////////////////////////////////////////////////////////////////////////////////////////
// Work-inefficient inclusive scan kernel
__global__ void workInefficient_inclusiveScan(uint* input, uint* output, uint n, uint* blockSums) {
    extern __shared__ uint shared[];
    int tid = threadIdx.x;
    int globalIndex = blockIdx.x * blockDim.x + tid;

    // Load input into shared memory, handling out-of-range indices
    if (globalIndex < n) {
        shared[tid] = input[globalIndex];
    }
    else {
        shared[tid] = 0; // Padding with zero
    }
    __syncthreads();

    // Perform a simple inclusive scan using shared memory
    for (uint stride = 1; stride < blockDim.x; stride *= 2) {
        uint temp = 0;
        if (tid >= stride) {
            temp = shared[tid - stride];
        }
        __syncthreads();
        shared[tid] += temp;
        __syncthreads();
    }

    // Write the results back to the output array
    if (globalIndex < n) {
        output[globalIndex] = shared[tid];
    }

    // Last thread in the block writes the block's sum to the blockSums array
    if (tid == blockDim.x - 1) {
        blockSums[blockIdx.x] = shared[blockDim.x - 1];
    }
}
 ////////////////////////////////////////////////////////////////////////////////////////////
// Work-efficient inclusive scan kernel
__global__ void workEfficient_inclusiveScan(uint* input, uint* output, uint n, uint* blockSums) {
    __shared__ uint shared[2 * 1024 * sizeof(uint)];
    int tid = threadIdx.x;
    int bIdx = blockIdx.x;
    int idx = bIdx * blockDim.x + tid;

    // Load input into shared memory
    shared[tid] = (idx < n) ? input[idx] : 0;
    __syncthreads();

    // Up-sweep (Reduction) phase
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = (tid + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            shared[index] += shared[index - stride];
        }
        __syncthreads();
    }

    // Down-sweep phase
    for (int stride = blockDim.x / 4; stride > 0; stride /= 2) {
        int index = (tid + 1) * 2 * stride - 1;
        if (index + stride < blockDim.x) {
            shared[index + stride] += shared[index];
        }
        __syncthreads();
    }

    // Write the processed data back to the output
    if (idx < n) {
        output[idx] = shared[tid];
    }

    // Last thread writes the last element to blockSums
    if (tid == blockDim.x - 1) {
        blockSums[bIdx] = shared[blockDim.x - 1];
    }
}

//////////////////////////////////////////////////////////////////////////////////////
__global__ void scanBlockSumsWithoutRec(uint* blockSums, int numBlocks) {
    extern __shared__ uint temp[];
    int tid = threadIdx.x;
    // Load block sums into shared memory
    if (tid < numBlocks) {
        temp[tid] = blockSums[tid];
    }
    else {
        temp[tid] = 0;
    }
    __syncthreads();

    // Inclusive scan using up-sweep
    for (unsigned int stride = 1; stride < numBlocks; stride *= 2) {
        //__syncthreads();
        int index = (tid + 1) * 2 * stride - 1;
        if (index < numBlocks) {//blockDim.x
            temp[index] += temp[index - stride];
        }
        __syncthreads();
        /*
        if (tid == 0) {
            printf("After up sweep stride %d: ", stride);
            for (int i = 0; i < numBlocks; i++) {
                printf("%d ", temp[i]);
            }
            printf("\n");
        }
        */
    }

    // Set the last element to zero to start the down-sweep
    if (tid == numBlocks - 1) {
        temp[numBlocks - 1] = 0;
    }
    __syncthreads();

    // Down-Sweep
    for (int stride = 1 << (31 - __clz(numBlocks - 1)); stride > 0; stride >>= 1) {
        int index = (tid + 1) * 2 * stride - 1;
        if (index + stride < numBlocks) {
            temp[index + stride] += temp[index];
        }
        __syncthreads();
        // Debug print after each stride
        /*
        if (tid == 0) {
            printf("After down sweep stride %d: ", stride);
            for (int i = 0; i < numBlocks; i++) {
                printf("%d ", temp[i]);
            }
            printf("\n");
        }
        */
    }
    // Write back to the block sums
    __syncthreads();

    if (tid < numBlocks) {
        blockSums[tid] = temp[tid];
    }
    //__syncthreads();
}
////////////////////////////////////////////////////////////////////////////////////////////
__global__ void addBlockSumsToOutput(uint* output, uint* blockSums, uint n, int numBlocks) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        if (blockIdx.x > 0) {
            output[idx] += blockSums[blockIdx.x - 1];
        }
    }
}
////////////////////////////////////////////////////////////////////////////////////////////
__global__ void scanBlockSumsVanilla(uint* blockSums, int numBlocks) {
    extern __shared__ uint temp[];
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;

    if (index < numBlocks) {
        temp[tid] = blockSums[index];
    }
    else {
        temp[tid] = 0;  // Important to avoid out-of-bounds access in shared memory
    }
    __syncthreads();

    // Perform up-sweep
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        int accessIndex = (tid + 1) * 2 * stride - 1;
        if (accessIndex < blockDim.x) {
            temp[accessIndex] += temp[accessIndex - stride];
        }
        __syncthreads();
    }

    // Down-Sweep
    for (int stride = 1 << (31 - __clz(blockDim.x - 1)); stride > 0; stride >>= 1) {
        int accessIndex = (tid + 1) * 2 * stride - 1;
        if (accessIndex + stride < blockDim.x) {
            temp[accessIndex + stride] += temp[accessIndex];
        }
        __syncthreads();
    }

    if (index < numBlocks) {
        blockSums[index] = temp[tid];
    }
}
//////////////////////////////////////////////////////////////////////////////////////
// Improved recursive function with enhanced error handling
hipError_t recursiveScanBlockSums(uint* blockSums, int numBlocks, uint* tempBuffer = nullptr) {
    bool isRootCall = (tempBuffer == nullptr);
    hipError_t cudaStatus;

    if (isRootCall) {
        size_t totalBufferSize = numBlocks * sizeof(uint);
        cudaStatus = hipMalloc((void**)&tempBuffer, totalBufferSize);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CUDA error: Failed to allocate temporary buffer: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }
    }

    if (numBlocks > 1) {
        int threads = min(numBlocks, MAX_THREADS_PER_BLOCK);
        int blocks = (numBlocks + threads - 1) / threads;
        uint* blockSumsNext = tempBuffer;

        scanBlockSumsVanilla <<<blocks, threads, threads * sizeof(uint) >>> (blockSums, numBlocks);
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CUDA Kernel error: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CUDA synchronization error: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }

        int nextNumBlocks = (numBlocks + threads - 1) / threads;
        cudaStatus = recursiveScanBlockSums(blockSumsNext, nextNumBlocks, tempBuffer + nextNumBlocks * sizeof(uint));
        if (cudaStatus != hipSuccess) {
            return cudaStatus;
        }
    }

    if (isRootCall) {
        hipFree(tempBuffer);
    }

    return hipSuccess;
}
//////////////////////////////////////////////////////////////////////////////////////
// CPU implementation of normal inclusive scan
void cpu_normal_inclusiveScan(uint* input, uint* output, uint n) {
    output[0] = input[0];
    for (uint i = 1; i < n; i++) {
        output[i] = output[i - 1] + input[i];
    }
}
//////////////////////////////////////////////////////////////////////////////////////
// Compare CPU and GPU results
bool compareResults(uint* cpuOutput, uint* gpuOutput, uint n) {
    for (uint i = 0; i < n; i++) {
        if (cpuOutput[i] != gpuOutput[i]) {
            printf("Mismatch at index %d: CPU=%d, GPU=%d\n", i, cpuOutput[i], gpuOutput[i]);
            return false;
        }
    }

    return true;
}
//////////////////////////////////////////////////////////////////////////////////////
float calculateGflops(int operations, float milliseconds) {
    return (operations / (milliseconds / 1000.0f)) / 1e9f; // Convert to seconds and then to Giga
}
//////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
    if (argc != 4 || (strcmp(argv[2], "-i") != 0)) {
        fprintf(stderr, "Usage: %s [scan-work-efficient | scan-work-inefficient] -i <vector_size>\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    uint n = atoi(argv[3]);
    if (n > 2 * MAX_THREADS_PER_BLOCK * MAX_BLOCKS) {
        fprintf(stderr, "Input size should be at most %d\n", 2 * MAX_THREADS_PER_BLOCK * MAX_BLOCKS);
        exit(EXIT_FAILURE);
    }

    uint* hostInput = (uint*)malloc(n * sizeof(uint));
    uint* hostOutput = (uint*)malloc(n * sizeof(uint));
    uint* cpuOutput = (uint*)malloc(n * sizeof(uint));
    uint* deviceInput;
    uint* deviceOutput;
    uint* deviceBlockSums;

    // Initialize input data
    srand(time(NULL));
    for (uint i = 0; i < n; i++) {
        hostInput[i] = rand() % 100;//1; // Simple input to demonstrate inclusive scan
    }

    // Allocate device memory
    allocateDeviceMemory(&deviceInput, n * sizeof(uint));
    allocateDeviceMemory(&deviceOutput, n * sizeof(uint));

    // Copy input data to device
    copyToDevice(deviceInput, hostInput, n * sizeof(uint));

    int threadsPerBlock = MAX_THREADS_PER_BLOCK;
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    allocateDeviceMemory(&deviceBlockSums, numBlocks * sizeof(uint));
    uint* hostBlockSums = (uint*)malloc(numBlocks * sizeof(uint));
    // Timing GPU execution
    // Initialize CUDA events
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float milliseconds = 0;
    if (strcmp(argv[1], "scan-work-inefficient") == 0) {
        // Start recording
        CUDA_CHECK(hipEventRecord(start, 0));
        workInefficient_inclusiveScan <<<numBlocks, threadsPerBlock, 4*threadsPerBlock * sizeof(uint) >>> (deviceInput, deviceOutput, n, deviceBlockSums);
        hipDeviceSynchronize();
        //scanBlockSumsWithoutRec <<< 1, numBlocks, numBlocks * sizeof(uint) >>> (deviceBlockSums, numBlocks);
        // Handle block sums recursively
        recursiveScanBlockSums(deviceBlockSums, numBlocks, nullptr);
        hipDeviceSynchronize();
        // Add block sums back to the output
        addBlockSumsToOutput <<<numBlocks, threadsPerBlock >>> (deviceOutput, deviceBlockSums, n, numBlocks);
        hipDeviceSynchronize();
        CUDA_CHECK(hipEventRecord(stop, 0));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        int operations = numBlocks * threadsPerBlock * (threadsPerBlock - 1);  // For work-inefficient
        cout << "Number of operations in work in-efficient: " << operations << endl;
        cout << "Number of numBlocks in work in-efficient: " << numBlocks << endl;
        float gflops = calculateGflops(operations, milliseconds);
        cout << "GPU execution time: " << milliseconds << " ms, GFLOPS: " << gflops << endl;
    }
    else if (strcmp(argv[1], "scan-work-efficient") == 0) {
        // Start recording
        CUDA_CHECK(hipEventRecord(start, 0));
        // Launch work-efficient scan
        workEfficient_inclusiveScan <<<numBlocks, threadsPerBlock >>> (deviceInput, deviceOutput, n, deviceBlockSums);
        hipDeviceSynchronize();

        // Handle block sums recursively
        recursiveScanBlockSums(deviceBlockSums, numBlocks, nullptr);
        //scanBlockSumsWithoutRec <<< 1, numBlocks, 2*numBlocks * sizeof(uint) >>> (deviceBlockSums, numBlocks);
        hipDeviceSynchronize();

        // Add block sums back to the output
        addBlockSumsToOutput <<<numBlocks, threadsPerBlock >>> (deviceOutput, deviceBlockSums, n, numBlocks);
        hipDeviceSynchronize();
        CUDA_CHECK(hipEventRecord(stop, 0));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        int operations = numBlocks * 2 * (threadsPerBlock - 1);  // For work-efficient
        cout <<"Number of operations in work efficient: " << operations << endl;
        cout << "Number of numBlocks in work in-efficient: " << numBlocks << endl;
        float gflops = calculateGflops(operations, milliseconds);
        cout << "GPU execution time: " << milliseconds << " ms, GFLOPS: " << gflops << endl;

    }
    else {
        fprintf(stderr, "Invalid kernel type. Choose either 'scan-work-efficient' or 'scan-work-inefficient'.\n");
        exit(EXIT_FAILURE);
    }
   
    
    // Calculate elapsed time
    cout << "GPU execution time: " << milliseconds << " ms\n";

    // Clean up
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    // Copy output data from device
    copyFromDevice(hostOutput, deviceOutput, n * sizeof(uint));

    // CPU inclusive scan for comparison
    auto start_cpu = chrono::high_resolution_clock::now();
    cpu_normal_inclusiveScan(hostInput, cpuOutput, n);
    auto end_cpu = chrono::high_resolution_clock::now();
    chrono::duration<double, milli> duration_cpu = end_cpu - start_cpu;
    printf("CPU execution time: %.3f ms\n", duration_cpu.count());

    // Compare CPU and GPU results
    bool matched = compareResults(cpuOutput, hostOutput, n);
    if (matched) {
        printf("CPU and GPU results match.\n");
    }
    else {
        printf("CPU and GPU results do not match.\n");
    }

    // Deallocate memory
    free(hostInput);
    free(hostOutput);
    free(cpuOutput);
    deallocateDeviceMemory(deviceInput);
    deallocateDeviceMemory(deviceOutput);
    deallocateDeviceMemory(deviceBlockSums);

    return 0;
}

